
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__ void add(float *d_in, float *d_out, int array_size){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(array_size%2==0 && idx == array_size-1)
        d_out[idx]=d_in[idx];
    else if(idx%2==0)
        d_out[idx/2] = d_in[idx] + d_in[idx+1];

}
int main(){
    
    //reading array size
    printf("Enter the size of array(less than 50k)\n");
    int array_size;
    scanf("%d",&array_size);
    printf("the sum of the first %d natural numbers is ",array_size);

    //allocating memory and generating the array
    float *h_in;
    h_in = (float *)malloc(array_size*sizeof(float));
    for(int i=0; i<array_size; i++){
        h_in[i]=i+1;
    }

    //allocating memory and copying data to device
    float *d_in, *d_out;
    int array_bytes = array_size*sizeof(float);
    hipMalloc((void**)&d_in, array_bytes);
    hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);

    //implementing performance metrics
    float time=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    while(array_size>1){

        float temptime=0;
        hipMalloc((void **)&d_out, array_bytes/2);
        hipEventRecord(start);
        if(array_size>512)
            add<<<(int)ceil(array_size/512.0f), 512>>>(d_in, d_out, array_size);
        else
            add<<<1, array_size>>>(d_in, d_out, array_size); 
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temptime, start, stop);
        time+=temptime;
        hipFree(d_in);
        d_in = d_out;
        array_size = (int)ceil(array_size/2.0f);

    }
    float res[1];
    hipMemcpy(res, d_in, sizeof(float), hipMemcpyDeviceToHost);
    printf("%f\n",res[0]);
    printf("time spent in gpu in ms : %f\n",time);
    return 0;
}