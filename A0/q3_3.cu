#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>

const int MAX_THREADS = 512;

__global__ void array_sum(float *d_in, float *d_sum){
    int ctr = 2;
    *d_sum=0;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    while(ctr!=MAX_THREADS*2){
        if(idx%ctr==0){
            d_in[idx]+=d_in[idx+ctr/2];
        }
        ctr*=2;
        __syncthreads;
    }
    atomicAdd(d_sum, d_in[blockDim.x*blockIdx.x]);
}
int main(){

    printf("enter a natural number\n");
    int n;
    scanf("%d",&n);
    int array_size = MAX_THREADS*ceil(n/MAX_THREADS);
    int array_bytes = array_size*sizeof(float);

    //generating input array
    float *h_in = (float *)malloc(array_bytes);
    for(int i=0;i<n;i++){
        h_in[i]=i+1;
    }
    h_in[0]=10;
    for(int i=n;i<array_size;i++){
        h_in[i]=0;
    }
    
    //copying data to device
    float *d_in;
    printf("working till here\n");
    hipMalloc((void **)&d_in, array_bytes);
    hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);

    //implementing performance metrics
    float time=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //calling the kernel
    float *d_sum;
    hipMalloc((void **)&d_sum, sizeof(float));
    hipEventRecord(start);
    array_sum<<<array_size/MAX_THREADS, MAX_THREADS>>>(d_in, d_sum);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("working\n");
    //copying answer from device to host
    float h_sum[1];
    hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    printf("sum of first %d natural numbers is %f\n",n ,h_sum[0]);
    printf("time spent in the gpu : %f\n",time);

    //freeing memory
    hipFree(d_in);
    hipFree(d_sum);

    return 0;






}