
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

const int MAX_THREADS = 512;

__global__ void array_sum(float *d_in, float *d_sum){
    int ctr = 2;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    while(ctr!=MAX_THREADS*2){
        if(idx%ctr==0){
            d_in[idx]+=d_in[idx+ctr/2];
        }
        ctr*=2;
        __syncthreads();
    }
    if(threadIdx.x==0)	    
	atomicAdd(d_sum, d_in[blockDim.x*blockIdx.x]);

}

int main(){

    printf("enter a natural number\n");
    int n;
    scanf("%d",&n);
    int array_size = MAX_THREADS*ceil((float)n/MAX_THREADS);
    int array_bytes = array_size*sizeof(float);

    //generating input array
    float *h_in = (float *)malloc(array_bytes);
    int i;
    for(i=0;i<n;i++){
        h_in[i]=i+1;
    }
    
    for(i=n;i<array_size;i++){
        h_in[i]=0;
    }


    //copying data to device
    float *d_in;
    hipMalloc((void **)&d_in, array_bytes);
    hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);

    //implementing performance metrics
    float time=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocating and initializing sum to zero    
    float *d_sum;
    float zero=0;
    hipMalloc((void **)&d_sum, sizeof(float));
    hipMemcpy(d_sum, &zero, sizeof(float), hipMemcpyHostToDevice);

    //calling the kernel
    hipEventRecord(start);
    array_sum<<<array_size/MAX_THREADS, MAX_THREADS>>>(d_in, d_sum);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //copying answer from device to host
    float h_sum[1];
    hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    
    printf("sum of first %d natural numbers is %f\n",n ,h_sum[0]);
    printf("time spent in the gpu : %f\n",time);

    //verifying correctness
    float sum = n*(n+1)/2.0f;
    if(sum==h_sum[0])
        printf("the calculated sum is correct\n");
    else 
        printf("the calculated sum is wrong\n");

    //freeing memory
    hipFree(d_in);
    hipFree(d_sum);

    return 0;
}
