
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

#define MAX_VAL 1e9

__global__ void compute(int *d_r, int *d_c, int *d_depth, int *max_depth, int *Q1, int *Q2, int nodes){
    int idx = threadIdx.x;
    __shared__ int len1, len2, curr_depth;
    int i;
    
    for(i=idx; i<nodes; i+=1024){
        d_depth[i] = MAX_VAL;
    }
    if(idx == 0){
        d_depth[0] = 0;
        curr_depth = 0;
        len1 = 1;
        len2 = 0;
        Q1[0] = 0;
    }
    __syncthreads();

    do{
        /*if(idx<len1){
            for(int j=d_r[Q1[idx]]; j<d_r[Q1[idx]+1]; j++){
                d_depth[d_c[j]] = curr_depth+1;
                Q2[len2++] = d_c[j];
            }
        }*/
        for(i=idx; i<nodes; i+=1024){
            if(i<len1){
                for(int j=d_r[Q1[i]]; j<d_r[Q1[i]+1]; j++){
                    if(atomicCAS(&d_depth[j], MAX_VAL, d_depth[i]+1) == MAX_VAL){
                        int t = atomicAdd(&len2,1);
                        Q2[t] = j;  
                    }
                }
            }
            else break;
        }
        __syncthreads();

        if(idx==0){
            for(i=0; i<len2; i++){
                Q1[i] = Q2[i];
            }
            len1 = len2;
            len2 = 0;
            curr_depth++;
        }
        __syncthreads();
    }while(len1);

    max_depth[0] = curr_depth-1;
}

int main(int argc, char *argv[]){
    if(argc<2){
        cout << "Usage: " << argv[0] << " <graph_file_name>\n";
        return 0;
    }

    ifstream input;
    input.open(argv[1]);

    int nodes, edges, i;
    input >> nodes;
    input >> edges;

    int *h_r = (int*)malloc((nodes+1)*sizeof(int));
    int *h_c = (int*)malloc(edges*2*sizeof(int));

    for(i=0; i<nodes+1; i++){
        input >> h_r[i];
    }
    for(i=0; i<edges*2; i++){
        input >> h_c[i];
    }
    
    int *Q1, *Q2, *d_r, *d_c, *d_depth, *max_depth;
    hipMalloc((void**)&Q1, nodes*sizeof(int));
    hipMalloc((void**)&Q2, nodes*sizeof(int));
    hipMalloc((void**)&d_r, (nodes+1)*sizeof(int));
    hipMalloc((void**)&d_c, edges*2*sizeof(int));
    hipMalloc((void**)&d_depth, nodes*sizeof(int));
    hipMalloc((void**)&max_depth, sizeof(int));

    hipMemcpy(d_r, h_r, (nodes+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, edges*2*sizeof(int), hipMemcpyHostToDevice);

    printf("Starting Computation\n");
    compute <<<1, 1024>>> (d_r, d_c, d_depth, max_depth, Q1, Q2, nodes);
    printf("Finished Computation\n");

    int *result = (int *)malloc(sizeof(int));
    printf("done\n");
    hipMemcpy(result, max_depth, sizeof(int), hipMemcpyDeviceToHost);

    printf("Depth : %d\n", result[0]);

    
    int *h_depth = (int*) malloc(nodes*sizeof(int));
	hipMemcpy(h_depth, d_depth, nodes*sizeof(int), hipMemcpyDeviceToHost);
	int *h_check_depth = (int*)malloc(nodes*sizeof(int));
	freopen(argv[2], "r", stdin);

	for(int i = 0; i < nodes; i++) {
		cin>>h_check_depth[i];
    }
    printf("Finished reading output file\n");
	bool flag = true;
	int count = 0;

    printf("Starting checking\n");
	for(int i = 0; i < nodes; i++) {
		if(h_depth[i] != h_check_depth[i]) {
            printf("Found %d, Expected %d\n",h_depth[i], h_check_depth[i]);
			flag = false;
			count++;
		}
    }
    printf("Finished checking\n");

	if(flag) {
		cout<<"Solution is correct!\n";
	}
	else {
		cout<<"Solution is incorrect!"<<endl;
		cout<<count<<" testcases failed.\n";
	}
    return 0;   
}