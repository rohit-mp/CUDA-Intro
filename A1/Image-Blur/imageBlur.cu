#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "wb.h"
using namespace std;

#define BLUR_SIZE 5
#define CHANNELS 3
#define CEIL(a, b) ((a-1)/b +1)

//@@ INSERT CODE HERE
__global__ void compute(float *deviceInputImageData, float *deviceOutputImageData, const int height, const int width){
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;
	for(int channel=0; channel<CHANNELS; channel++){
		float value=0,pixels=0;
		for(int i = xidx -BLUR_SIZE; i<=xidx + BLUR_SIZE; i++){
			for(int j = yidx -BLUR_SIZE; j<=yidx + BLUR_SIZE; j++){
				if(i>=0 && i<width && j>=0 && j<height){
					value += deviceInputImageData[CHANNELS*(j*width + i) + channel];
					pixels++;
				}
			}
		}
		value/=pixels;
		if(xidx<width && yidx<height)
			deviceOutputImageData[CHANNELS*(yidx*width+xidx) + channel] = value;
	}
}

int main(int argc, char *argv[]) {

	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;


	/* parse the input arguments */
	wbArg_t args = wbArg_read(argc, argv);
	inputImageFile = wbArg_getInputFile(args, 0);

	inputImage = wbImport(inputImageFile);

	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, CHANNELS);

	hostInputImageData  = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	// Allocate data
	wbTime_start(GPU, "Doing GPU memory allocation");
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * CHANNELS * sizeof(float));
	hipMalloc((void **)&deviceOutputImageData, imageWidth * imageHeight * CHANNELS * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	// Copy data
	wbTime_start(Copy, "Copying data to the GPU");
	hipMemcpy(deviceInputImageData, hostInputImageData,
		imageWidth * imageHeight * CHANNELS * sizeof(float), hipMemcpyHostToDevice);
	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");

	// Kernel call
	//@@ Insert Code here
	compute<<< dim3((int)CEIL(imageWidth, 32), (int)CEIL(imageHeight, 32), 1), dim3(32, 32, 1) >>>(deviceInputImageData, deviceOutputImageData, imageHeight, imageWidth);

	wbTime_stop(Compute, "Doing the computation on the GPU");

	// Copy data back
	wbTime_start(Copy, "Copying data from the GPU");
	hipMemcpy(hostOutputImageData, deviceOutputImageData,
		imageWidth * imageHeight * CHANNELS * sizeof(float), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	// Check solution
	wbSolution(args, outputImage);

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	wbImage_delete(outputImage);
	wbImage_delete(inputImage);
}
