#include "hip/hip_runtime.h"
#include "wb.h"
#include <bits/stdc++.h>
using namespace std;

#define NUM_BINS 128
#define CEIL(a, b) ((a-1)/b +1)

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);

		if (abort)
			exit(code);
	}
}

__global__ void compute(unsigned int *deviceInput, unsigned int *deviceBins, int *length){
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	__shared__ unsigned int hist[NUM_BINS];
	if(idx<128){
		hist[idx]=0;
	}
	__syncthreads();
	if(idx<length[0]){
		atomicAdd(&hist[deviceInput[idx]], 1);
	}
	__syncthreads();
	if(idx<128){
		atomicAdd(&deviceBins[idx], hist[idx]);
	}
}

int main(int argc, char *argv[]) {
	
	wbArg_t args;
	int inputLength;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;
	int *length;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImportChar(wbArg_getInputFile(args, 0), &inputLength);
	hostBins = (unsigned int *)calloc(NUM_BINS, sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	// Allocating GPU memory
	hipMalloc((void **)&deviceInput, inputLength * sizeof(uint));
	hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(uint));
	hipMalloc((void **)&length, sizeof(int));
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// Copying memory to the GPU
	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(deviceBins, hostBins, NUM_BINS * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(length, &inputLength, sizeof(int), hipMemcpyHostToDevice);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	compute<<< CEIL(inputLength, 1024), 1024 >>>(deviceInput, deviceBins, length);
	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");
	// Kernel computation
	
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// Copying the GPU memory back to the CPU
	hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(uint), hipMemcpyDeviceToHost);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// Freeing the GPU memory
	hipFree(deviceBins);
	hipFree(deviceInput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);
}
