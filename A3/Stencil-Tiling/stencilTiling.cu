#include "hip/hip_runtime.h"
#include "wb.h"
#include <bits/stdc++.h>
using namespace std;

#define CEIL(a, b) ((a-1)/b +1)
#define Clamp(a, start, end) (max(min(a, end), start))
#define value(arry, i, j, k) (arry[((i)*width + (j)) * depth + (k)])
#define output(i, j, k) value(output, i, j, k)
#define input(i, j, k) value(input, i, j, k)
#define data(i, j, k) data[i*121 + j*11 + k]

#define wbCheck(stmt)                                                           \
    do {                                                                        \
        hipError_t err = stmt;                                                 \
        if (err != hipSuccess) {                                               \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
            return -1;                                                          \
        }                                                                       \
    } while (0)

__global__ void compute(float *deviceInputData, float *deviceOutputData, int width, int height, int depth){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;    
    int i = x/depth;
    int j = x%depth;
    int k = y;

    __shared__ float shared_mem[ 34*34 ];

    float val = 0.1;
    if(i>=1 && i<height-1 && j>0 && j<width-1 && k>0 && k<depth-1){
        int a,b,c;
        for(a=-1; a<=1; a++){
            for(b=-1; b<=1; b++){
                for(c=-1; c<=1; c++){
                    shared_mem[((i+a)*width + (j+b)) * depth + (k+c)] = deviceInputData[((i+a)*width + (j+b)) * depth + (k+c)];
                }
            }
        }    
        __syncthreads();
        val = shared_mem[((i-1)*width + (j)) * depth + (k)] + shared_mem[((i)*width + (j-1)) * depth + (k)] + shared_mem[((i)*width + (j)) * depth + (k-1)]
            + shared_mem[((i+1)*width + (j)) * depth + (k)] + shared_mem[((i)*width + (j+1)) * depth + (k)] + shared_mem[((i)*width + (j)) * depth + (k+1)]
            - 6*shared_mem[((i)*width + (j)) * depth + (k)];
        val=0.1;
        deviceOutputData[((i)*width + (j)) * depth + (k)] = Clamp(val, 0.0, 1.0);
    }
    val=0.15;
    deviceOutputData[((i)*width + (j)) * depth + (k)] = Clamp(val, 0.0, 1.0);
}


static void launch_stencil(float *deviceOutputData, float *deviceInputData, 
    int width, int height, int depth) {
        compute<<< dim3( CEIL(width*depth, 32), CEIL(height, 32), 1), dim3(32,32,1) >>> (deviceInputData, deviceOutputData, width, height, depth);
    //Kernel call
}

int main(int argc, char *argv[]) {

    wbArg_t arg;
    int width;
    int height;
    int depth;
    char *inputFile;
    wbImage_t input;
    wbImage_t output;
    float *hostInputData;
    float *deviceInputData;
    float *deviceOutputData;

    arg = wbArg_read(argc, argv);

    inputFile = wbArg_getInputFile(arg, 0);
    input = wbImport(inputFile);

    width  = wbImage_getWidth(input);
    height = wbImage_getHeight(input);
    depth  = wbImage_getChannels(input);

    output = wbImage_new(width, height, depth);

    hostInputData  = wbImage_getData(input);

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **)&deviceInputData, width * height * depth * sizeof(float));
    hipMalloc((void **)&deviceOutputData, width * height * depth * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");

    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputData, hostInputData, width * height * depth * sizeof(float),
        hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");
    launch_stencil(deviceOutputData, deviceInputData, width, height, depth);
    wbTime_stop(Compute, "Doing the computation on the GPU");

    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(output.data, deviceOutputData, width * height * depth * sizeof(float),
        hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    /*float *hostOutputData = (float*)malloc(sizeof(float)*width*height*depth);
    hipMemcpy(hostOutputData, deviceOutputData, width*height*depth*sizeof(float), hipMemcpyDeviceToHost);
    int i,j,k;
    for(i=1; i<width-1; i++){
        for(j=1; j<height-1; j++){
            for(k=1; k<depth-1; k++){
                if(hostOutputData[((i)*width + (j)) * depth + (k)] != hostInputData[((i-1)*width + (j)) * depth + (k)] + hostInputData[((i)*width + (j-1)) * depth + (k)] + hostInputData[((i)*width + (j)) * depth + (k-1)]
                + hostInputData[((i+1)*width + (j)) * depth + (k)] + hostInputData[((i)*width + (j+1)) * depth + (k)] + hostInputData[((i)*width + (j)) * depth + (k+1)]
                - 6*hostInputData[((i)*width + (j)) * depth + (k)])
                printf("Failed at %d,%d,%d : Found:%f ; Expected:%f\n",i,j,k,hostOutputData[((i)*width + (j)) * depth + (k)], hostInputData[((i-1)*width + (j)) * depth + (k)] 
                + hostInputData[((i)*width + (j-1)) * depth + (k)] + hostInputData[((i)*width + (j)) * depth + (k-1)]
                + hostInputData[((i+1)*width + (j)) * depth + (k)] + hostInputData[((i)*width + (j+1)) * depth + (k)] + hostInputData[((i)*width + (j)) * depth + (k+1)]
                - 6*hostInputData[((i)*width + (j)) * depth + (k)]);
            }
        }
    }*/

    wbSolution(arg, output);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    wbImage_delete(output);
    wbImage_delete(input);
}