#include "hip/hip_runtime.h"
#include "wb.h"
#include <bits/stdc++.h>
using namespace std;

#define CEIL(a, b) ((a-1)/b +1)
#define Clamp(a, start, end) (max(min(a, end), start))
#define value(arry, i, j, k) (arry[((i)*width + (j)) * depth + (k)])
#define output(i, j, k) value(output, i, j, k)
#define input(i, j, k) value(input, i, j, k)
#define data(i, j, k) data[i*121 + j*11 + k]

#define wbCheck(stmt)                                                           \
    do {                                                                        \
        hipError_t err = stmt;                                                 \
        if (err != hipSuccess) {                                               \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
            return -1;                                                          \
        }                                                                       \
    } while (0)

__global__ void compute(float *deviceInputData, float *deviceOutputData, int width, int height, int depth){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int j = x/depth;
    int k = x%depth;
    int i = y;

    if(j>0 && j<width-1 && i>0 && i<height-1 && k>0 && k<depth-1){
        float val = deviceInputData[((i-1)*width + (j)) * depth + (k)] + deviceInputData[((i)*width + (j-1)) * depth + (k)] 
            + deviceInputData[((i)*width + (j)) * depth + (k-1)] + deviceInputData[((i+1)*width + (j)) * depth + (k)] 
            + deviceInputData[((i)*width + (j+1)) * depth + (k)] + deviceInputData[((i)*width + (j)) * depth + (k+1)]
            - 6*deviceInputData[((i)*width + (j)) * depth + (k)];
        deviceOutputData[((i)*width + (j)) * depth + (k)] = Clamp(val, 0.0, 1.0);
        if(i==1021) deviceOutputData[((i)*width + (j)) * depth + (k)] = 0.15;
    }
}

static void launch_stencil(float *deviceOutputData, float *deviceInputData, 
    int width, int height, int depth) {
        compute<<< dim3( CEIL(width*depth, 32), CEIL(height, 32), 1), dim3(32,32,1) >>> (deviceInputData, deviceOutputData, width, height, depth);
    //Kernel call
}

int main(int argc, char *argv[]) {

    wbArg_t arg;
    int width;
    int height;
    int depth;
    char *inputFile;
    wbImage_t input;
    wbImage_t output;
    float *hostInputData;
    float *deviceInputData;
    float *deviceOutputData;

    arg = wbArg_read(argc, argv);

    inputFile = wbArg_getInputFile(arg, 0);
    input = wbImport(inputFile);

    width  = wbImage_getWidth(input);
    height = wbImage_getHeight(input);
    depth  = wbImage_getChannels(input);

    output = wbImage_new(width, height, depth);

    hostInputData  = wbImage_getData(input);

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **)&deviceInputData, width * height * depth * sizeof(float));
    hipMalloc((void **)&deviceOutputData, width * height * depth * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");

    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputData, hostInputData, width * height * depth * sizeof(float),
        hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");
    launch_stencil(deviceOutputData, deviceInputData, width, height, depth);
    wbTime_stop(Compute, "Doing the computation on the GPU");

    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(output.data, deviceOutputData, width * height * depth * sizeof(float),
        hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    /*float *hostOutputData = (float*)malloc(sizeof(float)*width*height*depth);
    hipMemcpy(hostOutputData, deviceOutputData, sizeof(float)*width*height*depth, hipMemcpyDeviceToHost);
    int i,j,k;
    for(i=1; i<width-1; i++){
        for(j=1; j<height-1; j++){
            for(k=1; k<depth-1; k++){
                if(hostOutputData[((i)*width + (j)) * depth + (k)] != Clamp(hostInputData[((i-1)*width + (j)) * depth + (k)] + hostInputData[((i+1)*width + (j)) * depth + (k)]
                    + hostInputData[((i)*width + (j-1)) * depth + (k)] + hostInputData[((i)*width + (j+1)) * depth + (k)]
                    + hostInputData[((i)*width + (j)) * depth + (k-1)] + hostInputData[((i)*width + (j)) * depth + (k+1)]
                    - 6*hostInputData[((i)*width + (j)) * depth + (k)], 0.0, 1.0))
                    printf("%d,%d,%d Found:%f, Expected:%f\n",i,j,k,hostOutputData[((i)*width + (j)) * depth + (k)], hostInputData[((i-1)*width + (j)) * depth + (k)] + hostInputData[((i+1)*width + (j)) * depth + (k)]
                    + hostInputData[((i)*width + (j-1)) * depth + (k)] + hostInputData[((i)*width + (j+1)) * depth + (k)]
                    + hostInputData[((i)*width + (j)) * depth + (k-1)] + hostInputData[((i)*width + (j)) * depth + (k+1)]
                    - 6*hostInputData[((i)*width + (j)) * depth + (k)]);
            }
        }
    }*/

    wbSolution(arg, output);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    wbImage_delete(output);
    wbImage_delete(input);
}