
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

#define MAX_VAL 100000000

__global__ void compute(int *d_r, int *d_c, int *depth, int *max_depth, int nodes, int edges){
    int idx = threadIdx.x;
    int i;

    __shared__ int done;
    __shared__ int curr_depth;

    for(i=idx; i<nodes; i+=1024){
        depth[i] = MAX_VAL;
    }        
    if(idx==0){
        depth[0] = 0;
        curr_depth = 0;
        done=0;
    }
    __syncthreads();

    while(!done){
        if(idx == 0){
            done = 1;
        }
        __syncthreads();

        for(i=idx; i<nodes; i+=1024){
            if(depth[i] == curr_depth){
                done = 0;
                for(int j=d_r[i]; j<d_r[i+1]; j++){
                    int k = d_c[j];
                    if(depth[k] > curr_depth){
                        depth[k] = curr_depth+1;
                    }
                }
            }
        }
        if(idx==0 && done==0){
            curr_depth++;
        }
        __syncthreads();
    }
    if(idx == 0)
        *max_depth = curr_depth-1;
}

int main(int argc, char *argv[]){
    if(argc<2){
        cout << "Usage: " << argv[0] << " <graph_file_name>\n";
        return 0;
    }

    ifstream input;
    input.open(argv[1]);

    int nodes, edges, i;
    input >> nodes;
    input >> edges;

    // allocating host memory
    int *h_r = (int*)malloc((nodes+1)*sizeof(int));
    int *h_c = (int*)malloc(edges*2*sizeof(int));

    // reading inputs
    for(i=0; i<nodes+1; i++){
        input >> h_r[i];
    }
    for(i=0; i<edges*2; i++){
        input >> h_c[i];
    }

    // allocating device memory
    int *d_r, *d_c, *d_depth, *max_depth;
    hipMalloc((void**)&d_r, (nodes+1)*sizeof(int));
    hipMalloc((void**)&d_c, edges*2*sizeof(int));
    hipMalloc((void**)&d_depth, nodes*sizeof(int));
    hipMalloc((void**)&max_depth, sizeof(int));

    // copying data to device
    hipMemcpy(d_r, h_r, (nodes+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, edges*2*sizeof(int), hipMemcpyHostToDevice);
 
    // timer
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    hipEventRecord(start);

    // kernel call
    printf("Starting Computation\n");
    compute<<< 1,1024 >>> (d_r, d_c, d_depth, max_depth, nodes, edges);
    printf("Finished computation\n");

    // timer
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    cout<<"Compute time in GPU: "<<milliseconds<<"ms"<<endl;
    
    // copying results to host
    int *result = (int *)malloc(sizeof(int));
    hipMemcpy(result, max_depth, sizeof(int), hipMemcpyDeviceToHost);

    printf("Depth : %d\n", result[0]);

    // solution check
    int *h_depth = (int*) malloc(nodes*sizeof(int));
	hipMemcpy(h_depth, d_depth, nodes*sizeof(int), hipMemcpyDeviceToHost);
	int *h_check_depth = (int*)malloc(nodes*sizeof(int));
	freopen(argv[2], "r", stdin);

	for(int i = 0; i < nodes; i++) {
		cin>>h_check_depth[i];
	}
	bool flag = true;
	int count = 0;

	for(int i = 0; i < nodes; i++) {
		if(h_depth[i] != h_check_depth[i]) {
            printf("Found %d, Expected %d\n",h_depth[i], h_check_depth[i]);
			flag = false;
			count++;
		}
	}

	if(flag) {
		cout<<"Solution is correct!\n";
	}
	else {
		cout<<"Solution is incorrect!"<<endl;
		cout<<count<<" testcases failed.\n";
	}
}